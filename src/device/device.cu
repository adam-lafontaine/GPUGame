#include "device.hpp"
#include "cuda_def.cuh"

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cstring>

#endif


static void check_error(hipError_t err, cstr label = "")
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));

    if(std::strlen(label))
    {
        printf("\n%s", label);
    }
    
    printf("\n\n******************\n\n");

    #endif
}


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err, "cuda_memcpy_to_device");

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err, "cuda_memcpy_to_host");

    return err == hipSuccess;
}


bool cuda_no_errors(cstr label)
{
    hipError_t err = hipGetLastError();
    check_error(err, label);

    return err == hipSuccess;
}


bool cuda_launch_success(cstr label)
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err, label);

    return err == hipSuccess;
}





namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err, "malloc");

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool unified_malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if(!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMallocManaged((void**)&(buffer.data), n_bytes);
        check_error(err, "unified_malloc");

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if(buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err, "free");

            buffer.data = nullptr;

            return err == hipSuccess;
        }

        return true;
    }


    u8* push_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size < buffer.capacity);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;

        auto bytes_available = (buffer.capacity - buffer.size) >= n_bytes;
        assert(bytes_available);

        if(!is_valid || !bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if(is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }


    


    bool push_device_palette(MemoryBuffer& buffer, DeviceColorPalette& palette, u32 n_colors)
    {
        auto bytes_per_channel = sizeof(u8) * n_colors;
        size_t bytes_allocated = 0;

        for(u32 c = 0; c < RGB_CHANNELS; ++c)
        {
            auto data = push_bytes(buffer, bytes_per_channel);
            if(!data)
            {
                break;                
            }

            bytes_allocated += bytes_per_channel;
            palette.channels[c] = (u8*)data;
        }

        if(bytes_allocated == RGB_CHANNELS * bytes_per_channel)
        {
            palette.n_colors = n_colors;
            return true;
        }
        else if (bytes_allocated > 0)
        {
            pop_bytes(buffer, bytes_allocated);            
        }

        return false;
    }
}