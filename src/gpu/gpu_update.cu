#include "hip/hip_runtime.h"
#include "gpu_include.cuh"

#include <cassert>


constexpr auto N_PLAYER_WALL_COLLISIONS = N_PLAYER_ENTITIES * N_BROWN_ENTITIES;
constexpr auto N_BLUE_WALL_COLLISIONS = N_BLUE_ENTITIES * N_BROWN_ENTITIES;
constexpr auto N_PLAYER_BLUE_COLLISIONS =  N_PLAYER_ENTITIES * N_BLUE_ENTITIES;
constexpr auto N_BLUE_BLUE_COLLISIONS = N_BLUE_ENTITIES * N_BLUE_ENTITIES;


namespace gpuf
{
/*************************/



GPU_FUNCTION
static bool entity_will_intersect(Entity const& lhs, Entity const& rhs)
{
    auto delta_m = gpuf::sub_delta_m(lhs.next_position, rhs.next_position);
    
    auto rhs_rect = gpuf::make_rect(rhs.width_m, rhs.height_m);
    auto lhs_rect = gpuf::make_rect(delta_m, lhs.width_m, lhs.height_m);

    return gpuf::rect_intersect(lhs_rect, rhs_rect);
}


GPU_FUNCTION
static void move_player(Entity& entity, InputRecord const& input)
{    
    entity.dt = { 0.0f, 0.0f };

    if(input.input & INPUT::PLAYER_UP)
    {
        entity.dt.y -= input.est_dt_frame;
    }

    if(input.input & INPUT::PLAYER_DOWN)
    {
        entity.dt.y += input.est_dt_frame;
    }

    if(input.input & INPUT::PLAYER_LEFT)
    {
        entity.dt.x -= input.est_dt_frame;
    }

    if(input.input & INPUT::PLAYER_RIGHT)
    {
        entity.dt.x += input.est_dt_frame;
    }

    if(entity.dt.x != 0.0f && entity.dt.y != 0.0f)
    {
        entity.dt.x *= 0.707107f;
        entity.dt.y *= 0.707107f;
    }
}


GPU_FUNCTION 
void apply_current_input(Entity& entity, InputList const& inputs, u64 frame)
{
    entity.dt = { 0.0f, 0.0f };

    if(inputs.size == 0)
    {
        return;
    }

    auto& last = inputs.data[inputs.size - 1];

    auto is_last = last.frame_begin <= frame && frame < last.frame_end;

    if(!is_last)
    {
        return;
    }

    move_player(entity, last);
}


GPU_FUNCTION
static void stop_wall(Entity& ent, Entity const& wall)
{   
    if(!gpuf::is_active(ent) || !is_active(wall))
    {
        return;
    }

    auto delta = gpuf::sub_delta_m(ent.position, wall.position);
    
    auto w = gpuf::make_rect(wall.width_m, wall.height_m);
    auto e_start = gpuf::make_rect(delta, ent.width_m, ent.height_m);
    auto e_finish = gpuf::add_delta(e_start, ent.delta_pos_m);

    if(!gpuf::rect_intersect(e_finish, w))
    {
        return;
    }

    auto mm = 0.001f;

    auto e_x_finish = gpuf::add_delta(e_start, { ent.delta_pos_m.x, 0.0f });
    if(gpuf::rect_intersect(e_x_finish, w))
    {
        if(fabs(e_start.x_end - w.x_begin) < mm || fabs(e_start.x_begin - w.x_end) < mm)
        {
            ent.delta_pos_m.x = 0.0f;
        }
        else if(e_start.x_end < w.x_begin)
        {
            ent.delta_pos_m.x = w.x_begin - e_start.x_end - 0.5 * mm;
        }
        else if(e_start.x_begin > w.x_end)
        {
            ent.delta_pos_m.x = w.x_end - e_start.x_begin + 0.5 * mm;
        }
    }

    auto e_y_finish = gpuf::add_delta(e_start, { 0.0f, ent.delta_pos_m.y });
    if(gpuf::rect_intersect(e_y_finish, w))
    {
        if(fabs(e_start.y_end - w.y_begin) < mm || fabs(e_start.y_begin - w.y_end) < mm)
        {
            ent.delta_pos_m.y = 0.0f;
        }
        else if(e_start.y_end < w.y_begin)
        {
            ent.delta_pos_m.y = w.y_begin - e_start.y_end - 0.5 * mm;
        }
        else if(e_start.y_begin > w.y_end)
        {
            ent.delta_pos_m.y = w.y_end - e_start.y_begin + 0.5 * mm;
        }
    }
}


GPU_FUNCTION
static void bounce_wall(Entity& ent, Entity const& wall)
{
    if(!gpuf::is_active(ent) || !is_active(wall))
    {
        return;
    }

    if(ent.delta_pos_m.x == 0.0f && ent.delta_pos_m.y == 0.0f)
    {
        return;
    }

    auto delta = gpuf::sub_delta_m(ent.position, wall.position);
    
    auto w = gpuf::make_rect(wall.width_m, wall.height_m);
    auto e_start = gpuf::make_rect(delta, ent.width_m, ent.height_m);
    auto e_finish = gpuf::add_delta(e_start, ent.delta_pos_m);

    if(!gpuf::rect_intersect(e_finish, w))
    {
        return;
    }

    auto e_x_finish = gpuf::add_delta(e_start, { ent.delta_pos_m.x, 0.0f });
    if(gpuf::rect_intersect(e_x_finish, w))
    {
        ent.inv_x = true;
    }

    auto e_y_finish = gpuf::add_delta(e_start, { 0.0f, ent.delta_pos_m.y });
    if(gpuf::rect_intersect(e_y_finish, w))
    {
        ent.inv_y = true;
    }
    
}


GPU_FUNCTION
static void blue_blue(Entity& a, Entity const& b)
{
    if(!gpuf::is_active(a) || !is_active(b))
    {
        return;
    }

    if(a.delta_pos_m.x == 0.0f && a.delta_pos_m.y == 0.0f)
    {
        return;
    }

    auto delta = gpuf::sub_delta_m(a.position, b.next_position);
    
    auto b_finish = gpuf::make_rect(b.width_m, b.height_m);
    auto a_start = gpuf::make_rect(delta, a.width_m, a.height_m);
    auto a_finish = gpuf::add_delta(a_start, a.delta_pos_m);

    if(!gpuf::rect_intersect(a_finish, b_finish))
    {
        return;
    }

    auto a_x_finish = gpuf::add_delta(a_start, { a.delta_pos_m.x, 0.0f });
    if(gpuf::rect_intersect(a_x_finish, b_finish))
    {
        a.inv_x = true;
    }

    auto a_y_finish = gpuf::add_delta(a_start, { 0.0f, a.delta_pos_m.y });
    if(gpuf::rect_intersect(a_y_finish, b_finish))
    {
        a.inv_y = true;
    }
}


GPU_FUNCTION
static void player_blue(Entity const& player, Entity& blue)
{   
    if(!gpuf::is_active(player) || !gpuf::is_active(blue) || !gpuf::entity_will_intersect(player, blue))
    {
        return;
    }
    /*

    if(!gpu::equal(player.dt, { 0.0f, 0.0f }))
    {
        blue.dt = player.dt;
        blue.speed = player.speed;
        blue.delta_pos_m = player.delta_pos_m;
    }
    else
    {
        blue_blue(blue, player);
    }
    */

    //blue_blue(blue, player);
    gpuf::set_inactive(blue);
}


GPU_FUNCTION
static void entity_next_position(Entity& entity)
{
    if(!gpuf::is_active(entity))
    {
        return;
    }

    entity.delta_pos_m = gpuf::vec_mul(entity.dt, entity.speed);
    entity.next_position = gpuf::add_delta(entity.position, entity.delta_pos_m);
}


GPU_FUNCTION
static void update_entity_position(Entity& entity, ScreenProps const& props)
{
    if(entity.inv_x)
    {
        entity.delta_pos_m.x = 0.0f;
        entity.dt.x *= -1.0f;
    }

    if(entity.inv_y)
    {
        entity.delta_pos_m.y = 0.0f;
        entity.dt.y *= -1.0f;
    }

    entity.position = gpuf::add_delta(entity.position, entity.delta_pos_m);

    entity.next_position = entity.position;
    entity.delta_pos_m = { 0.0f, 0.0f };
    entity.inv_x = false;
    entity.inv_y = false;    
}


GPU_FUNCTION
static void update_entity_on_screen(Entity& entity, ScreenProps const& props)
{
    auto screen_width_m = props.screen_width_m;
    auto screen_height_m = props.screen_height_m;

    auto entity_screen_pos_m = gpuf::sub_delta_m(entity.position, props.screen_pos);
    auto entity_rect_m = gpuf::get_screen_rect(entity, entity_screen_pos_m);
    auto screen_rect_m = gpuf::make_rect(screen_width_m, screen_height_m);  

    auto is_onscreen = gpuf::rect_intersect(entity_rect_m, screen_rect_m);
    if(is_onscreen)
    {
        gpuf::set_onscreen(entity);
    }
    else
    {
        gpuf::set_offscreen(entity);
    }
}




/*************************/
}


GPU_KERNAL
static void gpu_next_movable_positions(DeviceMemory* device_p, UnifiedMemory* unified_p, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_MOVABLE_ENTITIES);

    auto& device = *device_p;
    auto& unified = *unified_p;

    auto offset = (u32)t;
    auto& entity = device.entities.data[offset];

    

    if(gpuf::is_player(entity.id))
    {
        if(entity.id == unified.user_player_entity_id)
        {
            gpuf::apply_current_input(entity, unified.current_inputs, unified.frame_count);
        }
        else
        {
            // previous input
        }        
    }
    
    gpuf::entity_next_position(entity);
}


GPU_KERNAL 
static void gpu_player_wall(DeviceMemory* device_p, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_PLAYER_WALL_COLLISIONS);

    auto& device = *device_p;

    auto offset = (u32)t;

    auto player_offset = offset / N_BROWN_ENTITIES;
    auto wall_offset = offset - player_offset * N_BROWN_ENTITIES;

    auto& player = device.player_entities.data[player_offset];
    auto& wall = device.wall_entities.data[wall_offset];

    gpuf::stop_wall(player, wall);
}


GPU_KERNAL
static void gpu_blue_wall(DeviceMemory* device_p, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_BLUE_WALL_COLLISIONS);

    auto& device = *device_p;

    auto offset = (u32)t;

    auto blue_offset = offset / N_BROWN_ENTITIES;
    auto wall_offset = offset - blue_offset * N_BROWN_ENTITIES;

    auto& blue = device.blue_entities.data[blue_offset];
    auto& wall = device.wall_entities.data[wall_offset];

    gpuf::bounce_wall(blue, wall);
}


GPU_KERNAL
static void gpu_player_blue(DeviceMemory* device_p, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_PLAYER_BLUE_COLLISIONS);

    auto& device = *device_p;

    auto offset = (u32)t;

    auto player_offset = offset / N_BLUE_ENTITIES;
    auto blue_offset = offset - player_offset * N_BLUE_ENTITIES;

    auto& blue = device.blue_entities.data[blue_offset];
    auto& player = device.player_entities.data[player_offset];

    gpuf::player_blue(player, blue);
}


GPU_KERNAL
static void gpu_blue_blue(DeviceMemory* device_p, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_BLUE_BLUE_COLLISIONS);

    auto& device = *device_p;

    auto offset = (u32)t;

    auto a_offset = offset / N_BLUE_ENTITIES;
    auto b_offset = offset - a_offset * N_BLUE_ENTITIES;

    if(a_offset == b_offset)
    {
        return;
    }

    auto& a = device.blue_entities.data[a_offset];
    auto& b = device.blue_entities.data[b_offset];

    gpuf::blue_blue(a, b);
}


GPU_KERNAL 
static void gpu_update_entity_positions(ScreenProps props, u32 n_threads)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t >= n_threads)
    {
        return;
    }

    assert(n_threads == N_ENTITIES);

    auto& device = *props.device_p;

    auto offset = (u32)t;

    auto& entity = device.entities.data[offset];
    if(!gpuf::is_active(entity))
    {
        return;
    }

    gpuf::update_entity_position(entity, props);
    gpuf::update_entity_on_screen(entity, props);
}


namespace gpu
{    
    void update(AppState& state)
    {        
        bool result = cuda::no_errors("gpu::update");
        assert(result);

        constexpr auto entity_threads = N_ENTITIES;
        constexpr auto entity_blocks = calc_thread_blocks(entity_threads);
        
        constexpr auto movable_threads = N_MOVABLE_ENTITIES;
        constexpr auto movable_blocks = calc_thread_blocks(movable_threads);

        constexpr auto player_wall_threads = N_PLAYER_WALL_COLLISIONS;
        constexpr auto player_wall_blocks = calc_thread_blocks(player_wall_threads);

        constexpr auto blue_wall_threads = N_BLUE_WALL_COLLISIONS;
        constexpr auto blue_wall_blocks = calc_thread_blocks(blue_wall_threads);

        constexpr auto player_blue_threads = N_PLAYER_BLUE_COLLISIONS;
        constexpr auto player_blue_blocks = calc_thread_blocks(player_blue_threads);

        constexpr auto blue_blue_threads = N_BLUE_BLUE_COLLISIONS;
        constexpr auto blue_blue_blocks = calc_thread_blocks(blue_blue_threads);

        auto device_p = state.device_buffer.data;
        auto unified_p = state.unified_buffer.data;
        
        cuda_launch_kernel(gpu_next_movable_positions, movable_blocks, THREADS_PER_BLOCK, device_p, unified_p, movable_threads);
        result = cuda::launch_success("gpu_next_movable_positions");
        assert(result);
        
        cuda_launch_kernel(gpu_player_wall, player_wall_blocks, THREADS_PER_BLOCK, device_p, player_wall_threads);
        result = cuda::launch_success("gpu_player_wall");
        assert(result);
        
        cuda_launch_kernel(gpu_blue_wall, blue_wall_blocks, THREADS_PER_BLOCK, device_p, blue_wall_threads);
        result = cuda::launch_success("gpu_blue_wall");
        assert(result);
        
        cuda_launch_kernel(gpu_player_blue, player_blue_blocks, THREADS_PER_BLOCK, device_p, player_blue_threads);
        result = cuda::launch_success("gpu_player_blue");
        assert(result);
        
        cuda_launch_kernel(gpu_blue_blue, blue_blue_blocks, THREADS_PER_BLOCK, device_p, blue_blue_threads);
        result = cuda::launch_success("gpu_blue_blue");
        assert(result);

        ScreenProps props{};
        props.device_p = state.device_buffer.data;
        props.screen_width_m = state.app_input.screen_width_m;
        props.screen_height_m = props.screen_width_m * state.screen_pixels.height / state.screen_pixels.width;
        props.screen_pos = state.app_input.screen_position;

        cuda_launch_kernel(gpu_update_entity_positions, entity_blocks, THREADS_PER_BLOCK, props, entity_threads);
        result = cuda::launch_success("gpu_update_entity_positions");
        assert(result);
        
    }
}